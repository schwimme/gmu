#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <opencv2/core/mat.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>

#include <math.h>
#include <string>
#include <stdio.h>


#define CUDA_VERIFY(fn, msg) \
do { \
	hipError_t err = (fn); \
	if (err != hipSuccess) { \
        fprintf(stderr, msg); \
		return err;\
	} \
} while (0)


__constant__ float c_gaussian[64];   //gaussian array in device side

// it uses only one axis of the kernel (1,2r) instead of a matrix (2r,2r)
inline hipError_t computeGaussianKernelCuda(const float delta, const int radius)
{
	float h_gaussian[64];
	for (int i = 0; i < 2 * radius + 1; ++i)
	{
		const float x = i - radius;
		h_gaussian[i] = expf(-(x * x) / (2.0f * delta * delta));
	}
	CUDA_VERIFY(hipMemcpyToSymbol(HIP_SYMBOL(c_gaussian), h_gaussian, sizeof(float)*(2 * radius + 1)), "CUDA Kernel Memcpy Host To Device Failed");
}

// it computes the euclidean distance between two points, each point a vector with 4 elements
__device__ inline float euclideanLenCuda(const float4 a, const float4 b, const float d)
{
	const float mod = (b.x - a.x) * (b.x - a.x) +
		(b.y - a.y) * (b.y - a.y) +
		(b.z - a.z) * (b.z - a.z) +
		(b.w - a.w) * (b.w - a.w);
	return expf(-mod / (2.0f * d * d));
}

__device__ inline float4 multiplyCuda(const float a, const float4 b)
{
	return{ a * b.x, a * b.y, a * b.z, a * b.w };
}

__device__ inline float4 addCuda(const float4 a, const float4 b)
{
	return{ a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w };
}

__global__ void bilateralFilterCudaKernel(const float4 * const d_input,
	float4 * const d_output,
	const float euclidean_delta,
	const int width, const int height,
	const int filter_radius)
{
	//2D Index of current thread
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if ((x<width) && (y<height))
	{
		float sum = 0.0f;
		float4 t = { 0.f, 0.f, 0.f, 0.f };
		const float4 center = d_input[y * width + x];
		const int r = filter_radius;

		float domainDist = 0.0f, colorDist = 0.0f, factor = 0.0f;

		for (int i = -r; i <= r; ++i)
		{
			int crtY = y + i; //clamp the neighbor pixel, prevent overflow
			if (crtY < 0)				crtY = 0;
			else if (crtY >= height)   	crtY = height - 1;

			for (int j = -r; j <= r; ++j)
			{
				int crtX = x + j;
				if (crtX < 0) 				crtX = 0;
				else if (crtX >= width)	 	crtX = width - 1;

				const float4 curPix = d_input[crtY * width + crtX];
				domainDist = c_gaussian[r + i] * c_gaussian[r + j];
				colorDist = euclideanLenCuda(curPix, center, euclidean_delta);
				factor = domainDist * colorDist;
				sum += factor;
				t = addCuda(t, multiplyCuda(factor, curPix));
			}
		}

		d_output[y * width + x] = multiplyCuda(1.f / sum, t);
	}
}

hipError_t bilateralFilterCuda(const float4 * const h_input,
	float4 * const h_output,
	const float euclidean_delta,
	const int width, const int height,
	const int filter_radius)
{
	// compute the gaussian kernel for the current radius and delta
	CUDA_VERIFY(computeGaussianKernelCuda(euclidean_delta, filter_radius), "Failed to compute gaussian kernel");

	// copy the input image from the CPU�s memory to the GPU�s global memory
	const int inputBytes = width * height * sizeof(float4);
	const int outputBytes = inputBytes;
	float4 *d_input, *d_output; // arrays in the GPU�s global memory
								// allocate device memory
	CUDA_VERIFY(hipMalloc<float4>(&d_input, inputBytes), "CUDA Malloc Failed");
	CUDA_VERIFY(hipMalloc<float4>(&d_output, outputBytes), "CUDA Malloc Failed");
	// copy data of input image to device memory
	CUDA_VERIFY(hipMemcpy(d_input, h_input, inputBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

	//GpuTimer timer;
	//timer.Start();

	// specify a reasonable grid and block sizes
	const dim3 block(16, 16);
	// calculate grid size to cover the whole image
	const dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	// launch the size conversion kernel
	bilateralFilterCudaKernel << <grid, block >> >(d_input, d_output, euclidean_delta, width, height, filter_radius);

	//timer.Stop();
	//printf("Own Cuda code ran in: %f msecs.\n", timer.Elapsed());

	// synchronize to check for any kernel launch errors
	CUDA_VERIFY(hipDeviceSynchronize(), "Kernel Launch Failed");

	//Copy back data from destination device meory to OpenCV output image
	CUDA_VERIFY(hipMemcpy(h_output, d_output, outputBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

	//Free the device memory
	CUDA_VERIFY(hipFree(d_input), "CUDA Free Failed");
	CUDA_VERIFY(hipFree(d_output), "CUDA Free Failed");
	//CUDA_VERIFY(hipDeviceReset(),"CUDA Device Reset Failed");

	return hipSuccess;
}


void processUsingCuda(std::string input_file, std::string output_file) {
	//Read input image from the disk
	cv::Mat input = cv::imread(input_file, cv::IMREAD_UNCHANGED);
	if (input.empty())
	{
		fprintf(stderr, "Image Not Found: %s", input_file.c_str());
		return;
	}

	// convert from char(0-255) BGR to float (0.0-0.1) RGBA
	cv::Mat inputRGBA;
	cv::cvtColor(input, inputRGBA, CV_BGR2RGBA, 4);
	inputRGBA.convertTo(inputRGBA, CV_32FC4);
	inputRGBA /= 255;

	//Create output image
	cv::Mat output(input.size(), inputRGBA.type());

	const float euclidean_delta = 3.0f;
	const int filter_radius = 3;

	bilateralFilterCuda((float4*)inputRGBA.ptr<float4>(),
		(float4*)output.ptr<float4>(),
		euclidean_delta,
		inputRGBA.cols, inputRGBA.rows,
		filter_radius);

	// convert back to char (0-255) BGR
	output *= 255;
	//output.convertTo(output, CV_8UC4);
	cvtColor(output, output, CV_RGBA2BGR, 3);

	imwrite(output_file, output);
}


int main(int argc, char** argv)
{
	if (argc < 3)
	{
		fprintf(stderr, "expected 2 params (path to input and output file)");
		return 1;
	}

	processUsingCuda(argv[1], argv[2]);

	return 0;
}
